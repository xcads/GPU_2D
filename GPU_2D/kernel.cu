#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "host_defines.h"
#include "hip/hip_runtime.h"
#include ""
#include "Header.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <algorithm>
#include <vector>
#include <time.h>
#include <string.h>
#include <limits.h>
#include <float.h>

using namespace std;
// PAIR structure
// VERTEX structure
struct Vertex {
	double x, y;
};

struct Pair {
	int index;
	int index1, index2;
	Line line1, line2;
	Vertex point;
	bool pruneFlag;
};

// Object Function
struct Objfunc {
	// xd = c1x + c2y
	double c1, c2;
};



#define FILENAME        Coefficient.txt
#define PI				3.14159265358979323846264338327950288419716939937510       
#define RANDOM_SEED     7
#define RANDOM_PARA     2000
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}



vector<struct Line> originalConstraints;
struct Vertex Solution;
int randomSeed = RANDOM_SEED;
//
//#define FILENAME        Coefficient.txt
//#define PI				3.14159265358979323846264338327950288419716939937510       
//#define RANDOM_SEED     7
//#define RANDOM_PARA     2000
//#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//
//
//#define HANDLE_NULL( a ) {if (a == NULL) { \
//                            printf( "Host memory failed in %s at line %d\n", \
//                                    __FILE__, __LINE__ ); \
//                            exit( EXIT_FAILURE );}}
//int randomSeed = RANDOM_SEED;
// LINE structure : Constraints
//typedef struct Line {
//	// a1x + a2y >= b
//	double a1, a2, b;
//	double slope;
//	bool pruneFlag;
//
//	int index;
//};

//// Object Function
//struct Objfunc {
//	// xd = c1x + c2y
//	double c1, c2;
//};

// VERTEX structure
//struct Vertex {
//	double x, y;
//};
//
//// PAIR structure
//struct Pair {
//	int index;
//	int index1, index2;
//	Line line1, line2;
//	Vertex point;
//	bool pruneFlag;
//};


 //vector<struct Line> originalConstraints;
 //struct Vertex Solution;
 //int randomSeed = RANDOM_SEED;

bool Intersection(struct Line *l1, struct Line *l2, struct Vertex *v1)
{
	if (fabs(l1->a1 * l2->a2 - l2->a1 * l1->a2) < DBL_EPSILON)
	{
		v1 = NULL;
		return false;
	}
	v1->x = -(l1->b * l2->a2 - l2->b * l1->a2) / (l1->a1 * l2->a2 - l2->a1 * l1->a2);
	v1->y = (l1->b * l2->a1 - l2->b * l1->a1) / (l1->a2 * l2->a1 - l1->a1 * l2->a2);
	return true;
}

//void Slope(struct Line *l)
//{
//	if (fabs(l->a2 - 0.0) < DBL_EPSILON)
//	{
//		if ((l->a1 > 0 && l->a2 < 0) || (l->a1 < 0 && l->a2 > 0))
//		{
//			l->slope = DBL_MAX;
//		}
//		else if ((l->a1 < 0 && l->a2 < 0) || (l->a1 > 0 && l->a2 > 0))
//		{
//			l->slope = -DBL_MAX;
//		}
//		else
//		{
//			l->slope = -l->a1 / l->a2;
//		}
//		return;
//	}
//	l->slope = -l->a1 / l->a2;
//	return;
//}

// Slope line
__device__ void Slope_d(struct Line *l)
{
	if (fabs(l->a2 - 0.0) < DBL_EPSILON)
	{
		if ((l->a1 > 0 && l->a2 < 0) || (l->a1 < 0 && l->a2 > 0))
		{
			l->slope = DBL_MAX;
		}
		else if ((l->a1 < 0 && l->a2 < 0) || (l->a1 > 0 && l->a2 > 0))
		{
			l->slope = -DBL_MAX;
		}
		else
		{
			l->slope = -l->a1 / l->a2;
		}
		return;
	}
	l->slope = -l->a1 / l->a2;
	return;
}

// Compare
//int cmp(const void *a, const void *b)
//{
//	struct Line *aa = (struct Line *)a;
//	struct Line *bb = (struct Line *)b;
//	return ((aa->slope > bb->slope) ? 1 : -1);
//}

// Rotation_d
__global__ void kRotation(struct Line oConstraints[], struct Line lines[], struct Objfunc *object, int *index, int *numG, int *numH)
{
	double thetaArc;

	if (object->c2 == 0 && object->c1 > 0) {
		thetaArc = -PI / 2;
	}
	else if (object->c2 == 0 && object->c1 < 0) {
		thetaArc = PI / 2;
	}
	else {
		thetaArc = atan(-object->c1 / object->c2);
	}

	int i;
	double a1Temp, a2Temp, bTemp;

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (offset < (*index)) {
		a1Temp = oConstraints[offset].a1;
		a2Temp = oConstraints[offset].a2;
		bTemp = oConstraints[offset].b;

		lines[offset].a1 = cos(thetaArc) * a1Temp + sin(thetaArc) * a2Temp;
		lines[offset].a2 = cos(thetaArc) * a2Temp - sin(thetaArc) * a1Temp;
		lines[offset].b = bTemp;
		lines[offset].index = offset;

		if (lines[offset].a2 > 0) {
			atomicAdd(numG, 1);
		}
		else if (lines[offset].a2 < 0) {
			atomicAdd(numH, 1);
		}
		else {
			return;
		}

		Slope_d(&lines[offset]);
		lines[offset].pruneFlag = true;
	}

	__syncthreads();
	__threadfence();

}

// Separation - O(n)
bool Separation(struct Line I1[], struct Line I2[], struct Line lines[], int numG, int numH)
{
	int index = numG + numH;
	int i, g = 0, h = 0;
	for (i = 0; i < index; i++) {
		if (lines[i].a2 > 0) {
			I1[g].a1 = -lines[i].a1 / lines[i].a2;
			I1[g].a2 = 1;
			I1[g].b = lines[i].b / lines[i].a2;
			Slope(&I1[g]);
			I1[g].slope = -I1[g].slope;
			I1[g].pruneFlag = true;
			I1[g].index = lines[i].index;

			g++;
		}
		else if (lines[i].a2 < 0) {
			I2[h].a1 = -lines[i].a1 / lines[i].a2;
			I2[h].a2 = 1;
			I2[h].b = lines[i].b / lines[i].a2;
			Slope(&I2[h]);
			I2[h].slope = -I2[h].slope;
			I2[h].pruneFlag = true;
			I2[h].index = lines[i].index;

			h++;
		}
		else {
			printf("%d %lf\n", i, lines[i].a2);
			return false;
		}
	}
	return true;
}

// Make pairs
bool MakePairs(struct Line I1[], struct Line I2[],
	struct Pair pairsG[], struct Pair pairsH[],
	int numG, int numH, int *index,
	double leftBound, double rightBound)
{
	int g, gtemp;
	(*index) = 0;
	for (g = 0; g < numG; g += 1) {
		// drop
		if (I1[g].pruneFlag == false) {
			continue;
		}
		for (gtemp = g + 1; gtemp < numG; gtemp++) {
			if (I1[gtemp].pruneFlag == true) {
				break;
			}
		}
		if (gtemp == numG) break;

		if (fabs(I1[g].slope - I1[gtemp].slope) < DBL_EPSILON) {
			if (I1[g].b > I1[gtemp].b) {
				I1[gtemp].pruneFlag = false;
				g = g - 1;
			}
			else {
				I1[g].pruneFlag = false;
				g = gtemp - 1;
			}

			continue;
		}
		struct Vertex *p = (struct Vertex *)malloc(sizeof(struct Vertex));
		Intersection(&I1[g], &I1[gtemp], p);
		if (p->x < leftBound || p->x > rightBound) {
			if (p->x < leftBound && (I1[g].slope > I1[gtemp].slope)) {
				I1[gtemp].pruneFlag = false;
				g = g - 1;
			}
			else if (p->x < leftBound && (I1[g].slope < I1[gtemp].slope)) {
				I1[g].pruneFlag = false;
				g = gtemp - 1;
			}
			else if (p->x > rightBound && (I1[g].slope < I1[gtemp].slope)) {
				I1[gtemp].pruneFlag = false;
				g = g - 1;
			}
			else if (p->x > rightBound && (I1[g].slope > I1[gtemp].slope)) {
				I1[g].pruneFlag = false;
				g = gtemp - 1;
			}
			continue;
		}
		pairsG[(*index)].index = (*index);
		pairsG[(*index)].line1 = I1[g];
		pairsG[(*index)].index1 = g;
		pairsG[(*index)].line2 = I1[gtemp];
		pairsG[(*index)].index2 = gtemp;
		pairsG[(*index)].point.x = p->x; pairsG[(*index)].point.y = p->y;

		(*index)++;
		g++;
	}

	return true;
}

// sg, Sg, sh, Sh
struct Vertex *TestingLine(struct Pair pairsG[], struct Pair pairsH[],
	struct Line I1[], struct Line I2[],
	int numG, int numH, int numDot,
	double *leftBound, double *rightBound)
{

	int index = (numDot == 0) ? 0 : (getRandomInt(&randomSeed, numDot));

	//printf("%d %d\n", index, numDot);


	if (numDot == 0) {
		int onlyOne = 0;
		bool isFeasible = false;
		struct Vertex *vSln = (struct Vertex *)malloc(sizeof(struct Vertex));
		vSln->y = -FLT_MAX;
		for (onlyOne = 0; onlyOne < numG; onlyOne++) {
			if (I1[onlyOne].pruneFlag == true) {
				isFeasible = true;
				break;
			}
		}
		if (isFeasible == true && numH != 0) {
			struct Vertex *vTemp = (struct Vertex *)malloc(sizeof(struct Vertex));
			for (int i = 0; i < numH; i++) {
				Intersection(&(I1[onlyOne]), &(I2[i]), vTemp);
				if (vSln->y < vTemp->y) {
					vSln->x = vTemp->x;
					vSln->y = vTemp->y;
				}
			}
			printf("sln: %lf %lf\n", vSln->x, vSln->y);
			return vSln;
		}
		else {
			/*
			for (int i = 0; i < numG; i++) {
			cout << "pruneFlag: " << I1[i].pruneFlag << endl;
			}*/
			cout << "No solution!\n";
			exit(0);
		}
	}

	//int index = round ? 1 : 0;
	double xPrimeG = pairsG[index].point.x;   // x' - xPrime
	double yPrimeG = pairsG[index].point.y;
	double yPrimeH;

	//cout << xPrimeG << '\n';

	// struct Line *sg = (&pairsG[index].line1.a1 < &pairsG[index].line2.a1) ? &pairsG[index].line1 : &pairsG[index].line2;
	// struct Line *Sg = (&pairsG[index].line1.a1 >= &pairsG[index].line2.a1) ? &pairsG[index].line1 : &pairsG[index].line2;
	struct Line *sg = NULL;
	struct Line *Sg = NULL;
	struct Line *sh = NULL;
	struct Line *Sh = NULL;
	// struct Line *sh = (&pairsH[index].line1.a1 < &pairsH[index].line2.a1) ? &pairsH[index].line1 : &pairsH[index].line2;
	// struct Line *Sh = (&pairsH[index].line1.a1 < &pairsH[index].line2.a1) ? &pairsH[index].line1 : &pairsH[index].line2;

	vector<int> linesG;
	vector<int> linesH;

	// Finding g(x') and H(x')
	for (int i = 0; i < numG; i++) {
		if (I1[i].pruneFlag == true) {
			if ((fabs(yPrimeG - (I1[i].a1 * xPrimeG + I1[i].b)) >DBL_EPSILON && yPrimeG < (I1[i].a1 * xPrimeG + I1[i].b)) || (sg == NULL || Sg == NULL)) {
				//printf("xPrime yPrime ???: %lf %lf %lf\n", xPrimeG, yPrimeG, (I1[i].a1 * xPrimeG + I1[i].b));



				yPrimeG = I1[i].a1 * xPrimeG + I1[i].b;
				sg = &I1[i];
				Sg = &I1[i];
			}
		}
	}
	for (int i = 0; i < numH; i++) {
		if (I2[i].pruneFlag == true) {
			if (sh == NULL || Sh == NULL) {
				sh = &I2[i];
				Sh = &I2[i];
				yPrimeH = I2[i].a1 * xPrimeG + I2[i].b;
			}
			else if (fabs(yPrimeH - (I2[i].a1 * xPrimeG + I2[i].b)) > DBL_EPSILON && yPrimeH > (I2[i].a1 * xPrimeG + I2[i].b)) {
				yPrimeH = I2[i].a1 * xPrimeG + I2[i].b;
				sh = &I2[i];
				Sh = &I2[i];
			}
		}
	}
	if (numH == 0) {
		yPrimeH = yPrimeG + 1000.0;
	}

	// Finding sg - min g(x') && Finding Sg - max g(x')
	/*
	struct Line *sg = &pairsG[0].line1;
	struct Line *Sg = &pairsG[0].line1;
	struct Line *sh = &pairsH[0].line1;
	struct Line *Sh = &pairsH[0].line1;
	*/
	for (int i = 0; i < numG; i++) {
		double currentLineValueG = I1[i].a1 * xPrimeG + I1[i].b;
		if (I1[i].pruneFlag == false || fabs(currentLineValueG - yPrimeG) >= DBL_EPSILON) {
			continue;
		}

		if (I1[i].a1 < sg->a1) {
			sg = &I1[i];
		}
		if (I1[i].a1 > Sg->a1) {
			Sg = &I1[i];
		}
	}
	// Finding sh - min h(x') && Finding Sh - max h(x')
	for (int i = 0; i < numH; i++) {
		double currentValueH = I2[i].a1 * xPrimeG + I2[i].b;
		if (I2[i].pruneFlag == false || fabs(currentValueH - yPrimeH) >= DBL_EPSILON) {
			continue;
		}

		if (I2[i].a1 < sh->a1) {
			sh = &I2[i];
		}
		if (I2[i].a1 > Sh->a1) {
			Sh = &I2[i];
		}
	}

	// Is feasible
	if (fabs(yPrimeG - yPrimeH) < DBL_EPSILON) {
		if (sg->a1 > 0 && sg->a1 >= Sh->a1) {
			// x* < x'
			if (sh != Sh) {
				sh->pruneFlag = false;
			}
			if (sg != Sg) {
				Sg->pruneFlag = false;
			}
			*rightBound = xPrimeG;
			//cout << "cccccccccc\n";
			return NULL;
		}
		else if (Sg->a1 < 0 && Sg->a1 <= sh->a1) {
			// x* > x'
			if (sh != Sh) {
				Sh->pruneFlag = false;
			}
			if (sg != Sg) {
				sg->pruneFlag = false;
			}
			*leftBound = xPrimeG;

			return NULL;
		}
		else {
			// x* = x'
			Solution.x = xPrimeG;
			Solution.y = yPrimeG;

			return &(Solution);
		}
	}
	else if (yPrimeG > yPrimeH) {   // infeasible
		if (sg->a1 > Sh->a1) {
			// x* < x'
			if (sh != Sh) {
				sh->pruneFlag = false;
			}
			if (sg != Sg) {
				Sg->pruneFlag = false;
			}

			else {
				if (pairsG[index].line1.a1 < pairsG[index].line2.a1) {
					I1[pairsG[index].index2].pruneFlag = false;
				}
				else if (pairsG[index].line1.a1 > pairsG[index].line2.a1) {
					I1[pairsG[index].index1].pruneFlag = false;
				}
			}
			*rightBound = xPrimeG;

			return NULL;
		}
		else if (Sg->a1 < sh->a1) {
			// x* > x'
			if (sh != Sh) {
				Sh->pruneFlag = false;
			}
			if (sg != Sg) {
				sg->pruneFlag = false;
			}

			else {
				if (pairsG[index].line1.a1 < pairsG[index].line2.a1) {
					I1[pairsG[index].index1].pruneFlag = false;
				}
				else if (pairsG[index].line1.a1 > pairsG[index].line2.a1) {

					I1[pairsG[index].index2].pruneFlag = false;
				}
			}
			*leftBound = xPrimeG;

			return NULL;
		}
		else if ((sg->a1 - Sh->a1) <= 0 && 0 <= (Sg->a1 - sh->a1)) {
			// no feasible
			printf("No feasible solution!\n");
			exit(0);
			return NULL;
		}
	}
	else if (yPrimeG < yPrimeH) {   // feasible
		if (sg->a1 > 0) {
			// x* < x'
			if (sg != Sg) {
				Sg->pruneFlag = false;
			}
			else {
				if (pairsG[index].line1.a1 < pairsG[index].line2.a1) {
					//pairsG[index].line2.pruneFlag = false;
					I1[pairsG[index].index2].pruneFlag = false;
				}
				else if (pairsG[index].line1.a1 > pairsG[index].line2.a1) {
					//pairsG[index].line1.pruneFlag = false;
					I1[pairsG[index].index1].pruneFlag = false;
				}
			}
			*rightBound = xPrimeG;
			//cout << "eeeeeeeeeeeeeeeee\n";
			return NULL;
		}
		else if (Sg->a1 < 0) {
			// x* > x'
			if (sg != Sg) {
				sg->pruneFlag = false;
			}
			else {
				if (pairsG[index].line1.a1 < pairsG[index].line2.a1) {
					//pairsG[index].line1.pruneFlag = false;
					I1[pairsG[index].index1].pruneFlag = false;
				}
				else if (pairsG[index].line1.a1 > pairsG[index].line2.a1) {

					I1[pairsG[index].index2].pruneFlag = false;
				}
			}
			*leftBound = xPrimeG;

			return NULL;
		}
		else if (sg->a1 <= 0 && 0 <= Sg->a1) {
			// x* = x'
			Solution.x = xPrimeG;
			Solution.y = yPrimeG;
			//cout << "hhhhhhhhhhhhhh\n";
			return &(Solution);
		}
	}
	return NULL;
}


///////////////////////////////////////////////////////////////////////////////////
static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}


bool Rotation_d(struct Line lines[], struct Objfunc object, int index, int *numG, int *numH)
{
	bool ret;

	// Original Constraints
	struct Line *dev_oConstraints;
	unsigned int size = index * sizeof(struct Line);

	HANDLE_ERROR(hipMalloc((void**)&dev_oConstraints, size));

	// Lines after Rotation_d
	struct Line *dev_lines;

	HANDLE_ERROR(hipMalloc((void**)&dev_lines, size));

	// Objective function
	struct Objfunc *dev_object;

	HANDLE_ERROR(hipMalloc((void**)&dev_object, sizeof(struct Objfunc)));

	// Numbers of lines
	int *dev_index;

	HANDLE_ERROR(hipMalloc((void**)&dev_index, sizeof(int)));

	// Num of G lines
	int *dev_numG;

	HANDLE_ERROR(hipMalloc((void**)&dev_numG, sizeof(int)));

	// Num of H lines
	int *dev_numH;

	HANDLE_ERROR(hipMalloc((void**)&dev_numH, sizeof(int)));

	// Space distribution
	unsigned int DIM = 1 + sqrt(index) / 16;

	dim3 blocks(DIM, DIM);
	dim3 threads(16, 16);

	(*numG) = (*numH) = 0;

	//float time_elapsed = 0;
	//hipEvent_t start, stop;
	//hipEventCreate(&start);    
	//hipEventCreate(&stop);

	//hipEventRecord(start, 0);  

	//hipEventRecord(stop, 0);    

	//hipEventSynchronize(start);   
	//hipEventSynchronize(stop);    
	//hipEventElapsedTime(&time_elapsed, start, stop);    




	// Copy from CPU to GPU
	HANDLE_ERROR(hipMemcpy(dev_oConstraints, &originalConstraints[0], size, hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(dev_object, &object, sizeof(struct Objfunc), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(dev_index, &index, sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(dev_numG, numG, sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(dev_numH, numH, sizeof(int), hipMemcpyHostToDevice));

	// Kernel function <<<blocks, threads>>>
	kRotation << <blocks, threads >> >(dev_oConstraints, dev_lines, dev_object, dev_index, dev_numG, dev_numH);

	// Copy from GPU to CPU
	HANDLE_ERROR(hipMemcpy(numG, dev_numG, sizeof(int), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipMemcpy(numH, dev_numH, sizeof(int), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipMemcpy(lines, dev_lines, size, hipMemcpyDeviceToHost));
	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	//printf("%f(ms)\n", time_elapsed);

	//printf("%d %d\n", (*numG), (*numH));

	if ((*numH) + (*numG) != index) {
		ret = false;
	}
	else {
		ret = true;
	}

	return ret;
}


void LinearProgramming(void)
{
	//float time_elapsed = 0;
	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);

	//hipEventRecord(start, 0);

	//hipEventRecord(stop, 0);

	//hipEventSynchronize(start);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&time_elapsed, start, stop);
	hipEvent_t start, stop;
	float elapsedTime = 0.0;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int indexRecord = 0;
	int numGRecord;
	int numHRecord;
	int indexPair;
	double leftBound, rightBound;
	double aTemp, bTemp, cTemp;
	bool judge = false;
	struct Objfunc object;

	//int round = 0;
	FILE* fp;

	fp = fopen("Coefficient.txt", "r");

	while (1) {
		fscanf_s(fp, "%lf%lf%lf", &aTemp, &bTemp, &cTemp);
		if (aTemp == 0.0 && bTemp == 0.0 && cTemp == 0.0) {
			break;
		}
		struct Line lineTemp;
		lineTemp.a1 = aTemp;
		lineTemp.a2 = bTemp;
		lineTemp.b = cTemp;
		originalConstraints.push_back(lineTemp);
		indexRecord++;
	}
	fscanf_s(fp, "%lf%lf", &object.c1, &object.c2);
	fscanf_s(fp, "%lf%lf", &leftBound, &rightBound);

	//cout << "lalala\n";

	struct Line *lines = (struct Line *)malloc(indexRecord * sizeof(struct Line));
	struct Line *I1 = (struct Line *)malloc(indexRecord * sizeof(struct Line));
	struct Line *I2 = (struct Line *)malloc(indexRecord * sizeof(struct Line));
	struct Pair *pairG = (struct Pair *)malloc(indexRecord * sizeof(struct Pair));
	struct Pair *pairH = (struct Pair *)malloc(indexRecord * sizeof(struct Pair));
	struct Vertex *sln = NULL;

	judge = Rotation_d(lines, object, indexRecord, &numGRecord, &numHRecord);
	if (judge == false) {
		printf("Fatal Error at LinearProgramming() - Rotation_d()!\n");
		exit(-1);
	}

	judge = Separation(I1, I2, lines, numGRecord, numHRecord);
	if (judge == false) {
		printf("Fatal Error at LinearProgramming() - Segmentation()!\n");
		exit(-1);
	}

	//cout << numGRecord << " " << numHRecord << '\n';
	/*
	for (int i = 0; i < numGRecord; I++) {
	printf("")
	}
	*/

	while (1) {
		judge = MakePairs(I1, I2, pairG, pairH, numGRecord, numHRecord, &indexPair, leftBound, rightBound);
		if (judge == false) {
			printf("Fatal Error at LinearProgramming() - MakePairs()!\n");
			exit(-1);
		}

		sln = TestingLine(pairG, pairH, I1, I2, numGRecord, numHRecord, indexPair, &leftBound, &rightBound);
		//cout << leftBound << " " << rightBound << '\n';
		if (sln != NULL) {
			break;
		}
	}
	//printf("sln: %lf %lf\n", sln->x, sln->y);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << elapsedTime << endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);
	fclose(fp);
	return;
}


int main()
{
	LinearProgramming();

	return 0;
}

